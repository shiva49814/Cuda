
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>

#include <stdio.h>


int THREADS_PER_BLOCK;
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}


__global__ void polynomial(float* array, float* poly, int degree, int n) {
  
  int index = threadIdx.x+ blockIdx.x* blockDim.x;
  if(index < n){
  float out = 0.;
  float xtothepowerof = 1.;
  for (int i=0; i<=degree; i++) {
    out += xtothepowerof * poly[i];
    xtothepowerof *= array[index];
  }
  array[index] = out;
  }
}

void polynomial_expansion (float* poly, int degree,
			   int n, float* array) {
 float *d_poly, *d_array;
 
 int size_array = n * sizeof(float);
 int size_poly = (degree+1) * sizeof(float);
 

//std::cout<<1<<std::endl;
 //Allocating memory on the GPU.
  HANDLE_ERROR(hipMalloc(&d_array, size_array));  //std::cout<<2<<std::endl;

  HANDLE_ERROR(hipMalloc(&d_poly, size_poly));  //std::cout<<3<<std::endl;


 //Copying variables from cpu to gpu.
 HANDLE_ERROR(hipMemcpy(d_poly, poly, size_poly, hipMemcpyHostToDevice)); 
 HANDLE_ERROR(hipMemcpy(d_array, array, size_array, hipMemcpyHostToDevice)); 


  // Launch add() kernel on GPU
  polynomial<<<(n+ THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_array,d_poly, degree, n);

// Copy result back to host

  HANDLE_ERROR(hipMemcpy(array, d_array, size_array, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(poly, d_poly, size_poly, hipMemcpyDeviceToHost));

 
 //Cleanup
 HANDLE_ERROR(hipFree(d_array)); 
 HANDLE_ERROR(hipFree(d_poly)); 

}


int main (int argc, char* argv[]) {
  //TODO: add usage
  
  if (argc < 4) {
     std::cerr<<"usage: "<<argv[0]<<" n degree blocksize"<<std::endl;
     return -1;
  }

  int n = atoi(argv[1]); //TODO: atoi is an unsafe function
  int degree = atoi(argv[2]);
  THREADS_PER_BLOCK= atoi(argv[3]);
  int nbiter = 1;

  float* array = new float[n];
  float* poly = new float[degree+1];
  for (int i=0; i<n; ++i)
    array[i] = 1.;

  for (int i=0; i<degree+1; ++i)
    poly[i] = 1.;

 

  std::chrono::time_point<std::chrono::system_clock> begin, end;
  begin = std::chrono::system_clock::now();

 for (int iter = 0; iter<nbiter; ++iter)
    polynomial_expansion (poly, degree, n, array);



  end = std::chrono::system_clock::now();
  std::chrono::duration<double> totaltime = (end-begin)/nbiter;

  std::cerr<<array[0]<<std::endl;
  std::cout<<n<<" "<<degree<<" "<<totaltime.count()<<std::endl;


  delete[] array;
  delete[] poly;

  return 0;
}
